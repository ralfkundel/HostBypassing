#include "hip/hip_runtime.h"
//Authors: Leonard Anderweit, Ralf Kundel
//2022

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <fcntl.h>
#include <sys/ioctl.h>
#include <unistd.h>


#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "dpdk.h"
#include "../settings.h"

#define PIN_MEM     _IOW('a',0,struct ioctl_args*)
#define UNPIN_MEM   _IOW('a',1,void**)
#define RD_ADDR     _IOR('a',2,void**)

#define IXGBE_ADV_TX_DESC_DTYP_DATA 3<<20
#define IXGBE_ADV_TX_DESC_DCMD_EOP 1<<24
#define IXGBE_ADV_TX_DESC_DCMD_INS_FCS 1<<25
#define IXGBE_ADV_TX_DESC_DCMD_RS 1<<27
#define IXGBE_ADV_TX_DESC_DCMD_ADVD 1<<29
#define IXGBE_ADV_TX_PAYLEN_SHIFT 14


struct ioctl_args {
    uint64_t vaddr;
    uint64_t size;
    uint32_t bus;
    uint32_t devfn;
};

struct pkt_info {
    uint16_t position; //within the packet buffer mem
    uint16_t length; //in bytes
};


__device__ volatile pkt_info malloc_empty_desc[PKT_BUFFER_SIZE*RINGS];
__device__ volatile uint32_t malloc_empty_desc_head[RINGS];
__shared__ uint32_t malloc_empty_desc_tail[RINGS];
//__shared__ memory can only be shared within a kernel

__device__ volatile pkt_info malloc_received_desc[PKT_BUFFER_SIZE*RINGS];
__device__ volatile uint32_t malloc_received_desc_head[RINGS];
__shared__ uint32_t malloc_received_desc_tail[RINGS];


__global__ void
init_empty_desc(){
    for(int i = 0; i<PKT_BUFFER_SIZE*RINGS; i++){
        malloc_empty_desc[i].position = i;
        malloc_empty_desc[i].length = 0;
    }
}


__global__ void
receive(uint64_t *rx_desc_base_virt, uint32_t* rdt_reg){ // rdt receive descriptor tail
    int index = threadIdx.x; // receive ring separator
    
    uint16_t rx_desc_cp[RX_RING_SIZE]; //local copy of mem address in rings
    
    //initialize
    malloc_received_desc_head[index] = 0;
    
    int buf_offset = index * PKT_BUFFER_SIZE;
    volatile union ixgbe_adv_rx_desc* desc_mem = (volatile union ixgbe_adv_rx_desc*) (rx_desc_base_virt + index * RX_RING_SIZE * DESC_SIZE/8); //RX_RING_SIZE ==256, DESC_SIZE==16
    uint16_t pos;
    malloc_empty_desc_tail[index] = 1;
    
	for(uint32_t i = 0; i<RX_RING_SIZE;i++){ //init the first RX_RING_SIZE descriptors for receiving
        pos = malloc_empty_desc[i+buf_offset].position;
		desc_mem[i].read.pkt_addr = GPU_PKT_BUFFER_MEM_ADDR + MEM_PER_PKT * pos;
		desc_mem[i].read.hdr_addr = 0;
        rx_desc_cp[i] = pos;
        malloc_empty_desc_tail[index]++;
	}
	
	uint32_t counter[8];
    
    //end initialize
    

    volatile union ixgbe_adv_rx_desc *rx_ring = (volatile union ixgbe_adv_rx_desc* ) (rx_desc_base_virt + index * RX_RING_SIZE * DESC_SIZE/8);
	volatile union ixgbe_adv_rx_desc *rx_desc;
	uint32_t staterr;
    uint16_t new_pos;
    uint16_t length;
    uint32_t rx_pkt_index = 0;
	
	while(true){
		rx_desc = &rx_ring[rx_pkt_index];
	    staterr = rx_desc->wb.upper.status_error;
	    if(staterr&1) { //check for DD bit
            length = rx_desc->wb.upper.length;
            #if DEBUG
            printf("index%d checking pkt at:%u len:%u\n", index, rx_pkt_index, length);
            #endif
            if(length>0){ // new packet
                #if DEBUG
                printf("index %d: new pkt at rx_pkt_index: %u\n", index,rx_pkt_index);
                printf("index%d head %u, tail %u\n", index, malloc_received_desc_head[index], malloc_received_desc_tail[index]);
                printf("empty_desc_tail[index] %d\n", malloc_empty_desc_tail[index]);
                
                #endif
            
                
                if(malloc_empty_desc_tail[index] != malloc_empty_desc_head[index]){ // check for new empty memory
                
                    counter[index]++;
                
                    malloc_received_desc[malloc_received_desc_head[index]+buf_offset].position = rx_desc_cp[rx_pkt_index];
                    malloc_received_desc[malloc_received_desc_head[index]+buf_offset].length = length;
                    malloc_received_desc_head[index] = (malloc_received_desc_head[index] >= PKT_BUFFER_SIZE-1)? 0 : malloc_received_desc_head[index]+1;
                    // write new desc
                    new_pos = malloc_empty_desc[malloc_empty_desc_tail[index]+buf_offset].position;
		            rx_desc->read.hdr_addr = 0;
		            rx_desc->read.pkt_addr = GPU_PKT_BUFFER_MEM_ADDR + MEM_PER_PKT * new_pos;
                    rx_desc_cp[rx_pkt_index] = new_pos;
                    malloc_empty_desc_tail[index] = (malloc_empty_desc_tail[index] >= PKT_BUFFER_SIZE-1)? 0 : malloc_empty_desc_tail[index]+1;
                    rdt_reg[index*NIC_POINTER_OFFS/4] = rx_pkt_index;
                    rx_pkt_index = (rx_pkt_index >= RX_RING_SIZE-1)? 0 : rx_pkt_index+1;
                    //printf("index %d\n", index);
                    
                    #if DEBUG
                    if (counter[index]%10000 == 0){
                        printf("counter index: %d, counter: %d \n", index, counter[index]);
                    }
                    #endif
                } else{
                    // return;
                    #if DEBUG
                    printf("shit; no mem\n");
                    #endif
                    printf("shit; no mem\n");
                    continue;
                }
            
            } //end length > 0
            
        }
        #if DEBUG
        printf("index%d rx_pkt_index: %u\n", index,rx_pkt_index);
        #endif
    }
    
}

__global__ void
send(uint64_t *tx_desc_base_virt, uint32_t* tdt_reg){ // tdt transmit descriptor tail
    int index = threadIdx.x;
    
    /* initialize */
    uint32_t tx_pkt_index = 0;
    malloc_received_desc_tail[index] = 0;
    uint16_t tx_desc_cp[RX_RING_SIZE]; //local copy of mem address in rings
    
    int buf_offset = index * PKT_BUFFER_SIZE;
    volatile union ixgbe_adv_tx_desc* tx_desc_ring = (volatile union ixgbe_adv_tx_desc*) (tx_desc_base_virt + index * TX_RING_SIZE * DESC_SIZE/8);
    
    malloc_empty_desc_head[index] = PKT_BUFFER_SIZE-1;
        
    for(int i = 0; i<TX_RING_SIZE; i++){
        tx_desc_ring[i].wb.rsvd = 0;
        tx_desc_ring[i].wb.nxtseq_seed = 0;
        tx_desc_ring[i].wb.status = 1;
        tx_desc_cp[i] = malloc_empty_desc[i+buf_offset+PKT_BUFFER_SIZE-TX_RING_SIZE].position;
    }

    /* end initialize */
    
    

    uint16_t pkt_len;
    uint16_t new_pos;
    
    while(true)
    if(malloc_received_desc_head[index] != malloc_received_desc_tail[index]){
        #if DEBUG
        printf("index%d nxt %x, stat %x at %u\n", index, tx_desc_ring[tx_pkt_index].wb.nxtseq_seed, tx_desc_ring[tx_pkt_index].wb.status, tx_pkt_index);
        #endif
        #if WB
        if(tx_desc_ring[tx_pkt_index].wb.status & 1){
        #endif
            #if DEBUG
            printf("index%d send pkt %u\n", index, tx_pkt_index);
            #endif
        
            malloc_empty_desc[malloc_empty_desc_head[index]+buf_offset].position = tx_desc_cp[tx_pkt_index];
            malloc_empty_desc[malloc_empty_desc_head[index]+buf_offset].length = 0;
        
            pkt_len = malloc_received_desc[malloc_received_desc_tail[index]+buf_offset].length;
            new_pos = malloc_received_desc[malloc_received_desc_tail[index]+buf_offset].position;
            tx_desc_ring[tx_pkt_index].read.buffer_addr   = GPU_PKT_BUFFER_MEM_ADDR + MEM_PER_PKT * new_pos;
            #if WB
            tx_desc_ring[tx_pkt_index].read.cmd_type_len  = (pkt_len) | IXGBE_ADV_TX_DESC_DTYP_DATA | IXGBE_ADV_TX_DESC_DCMD_ADVD | IXGBE_ADV_TX_DESC_DCMD_EOP | IXGBE_ADV_TX_DESC_DCMD_INS_FCS | IXGBE_ADV_TX_DESC_DCMD_RS;
            #else
            tx_desc_ring[tx_pkt_index].read.cmd_type_len  = (pkt_len) | IXGBE_ADV_TX_DESC_DTYP_DATA | IXGBE_ADV_TX_DESC_DCMD_ADVD | IXGBE_ADV_TX_DESC_DCMD_EOP | IXGBE_ADV_TX_DESC_DCMD_INS_FCS;
            #endif
            tx_desc_ring[tx_pkt_index].read.olinfo_status = (pkt_len) << IXGBE_ADV_TX_PAYLEN_SHIFT;
            tx_desc_cp[tx_pkt_index] = new_pos;
            #if DEBUG
            printf("index%d nxt %x, stat %x at %u\n", index, tx_desc_ring[tx_pkt_index].wb.nxtseq_seed, tx_desc_ring[tx_pkt_index].wb.status, tx_pkt_index);
            #endif

            // increase tx tail pointer
            tx_pkt_index = (tx_pkt_index >= TX_RING_SIZE-1)? 0 : tx_pkt_index+1;
            //__threadfence_block(); --> crashes when multiple rings
            tdt_reg[index*NIC_POINTER_OFFS/4] = tx_pkt_index; // tail in nic
            malloc_received_desc_tail[index] = (malloc_received_desc_tail[index] >= PKT_BUFFER_SIZE-1)? 0 : malloc_received_desc_tail[index]+1;
            malloc_empty_desc_head[index] = (malloc_empty_desc_head[index] >= PKT_BUFFER_SIZE-1)? 0 : malloc_empty_desc_head[index]+1;
        #if WB
        }else{
            printf("break send\n");
            continue;
        }
        #endif
    }
}


int pin_mem(uint64_t address, uint64_t size){
    int fd;
    fd = open("/dev/etx_device", O_RDWR);
    if(fd < 0) {
        printf("Cannot open device file...\n");
        return -1;
    }
    struct ioctl_args args;
    args.vaddr = address;
    args.size = size;
    args.bus = NIC_BUS;
    args.devfn = NIC_DEVFN;
    ioctl(fd, PIN_MEM, &args);
    close(fd);
    return 0 ;
}

int unpin_mem(uint64_t address){
    int fd;
    fd = open("/dev/etx_device", O_RDWR);
    if(fd < 0) {
        printf("Cannot open device file...\n");
        return -1;
    }
    ioctl(fd, UNPIN_MEM, 0);
    close(fd);
    return 0 ;
}

int* init_gpu(){

    int *d_pointer;
    //extern __shared__ int tmp[MEM_SIZE/4];  //shared memory cannot be pinned
    //d_pointer = tmp;
    hipMalloc((void **)&d_pointer, MEM_SIZE);
    hipPointerAttribute_t attrs;
    hipPointerGetAttributes(&attrs, d_pointer);
    unsigned int flag = 1;
    hipError_t status = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t)attrs.devicePointer);
    pin_mem((uint64_t) attrs.devicePointer, MEM_SIZE);
    return d_pointer;
}


int main(int argc, char *argv[]){
    int deviceId = 0; //1; //TODO dirty, if multiple GPUs are in a single system, this must be adapted manually
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    printf("pciDeviceID: %x \n", deviceProp.pciDeviceID);
    printf("pciBusID: %x \n", deviceProp.pciBusID);
    printf("pciDomainID: %x \n", deviceProp.pciDomainID);
    hipDeviceReset();
    hipSetDevice(deviceId);
    hipError_t err;
    uint32_t* rdt_reg;
    uint32_t* tdt_reg;

    // make nic tailpointer accessible for gpu
    int fd = open("/dev/mem",O_RDWR);
    if(fd<0) {
		printf("couldn't open mem resource\n");
		return -1;
	}
	void* mem = mmap(NULL, NIC_REG_SIZE, PROT_READ | PROT_WRITE, MAP_SHARED, fd, NIC_REG_ADDR);
	close(fd);
	if( mem==MAP_FAILED){
		printf("mmap failed errno:%s\n",strerror(errno));
		return -1;
	}

    rdt_reg = (uint32_t*) mem + (NIC_RDT_OFFS)/4;
    tdt_reg = (uint32_t*) mem + (NIC_TDT_OFFS)/4;

    err = hipHostRegister((void*)rdt_reg,512,hipHostRegisterIoMemory);
    if(err!=hipSuccess){
        printf("hostRegister failed!! err:%d\n",err);
    }
    err = hipHostRegister((void*)tdt_reg,512,hipHostRegisterIoMemory);
    if(err!=hipSuccess){
        printf("hostRegister failed!! err:%d\n",err);
    }

    int ret;
    hipDeviceGetAttribute(&ret, hipDeviceAttributeCanUseHostPointerForRegisteredMem, 0);
    printf("hipDeviceAttributeCanUseHostPointerForRegisteredMem: %d\n",ret); // needs to be 1 for code to work

    void *d_pointer = init_gpu(); // virtuelle adresse gpu memory

    static uint64_t* rx_desc_base_virt = (uint64_t*) d_pointer;
    static uint64_t* tx_desc_base_virt = (uint64_t*) d_pointer + 8*4096/8; //4096 byte per ring, up to 8 rx rings

    printf("RINGS: %d\n",RINGS);

    init_empty_desc<<<1,1>>>();
    hipDeviceSynchronize();
    err = hipGetLastError();
    if(err!=hipSuccess){
        printf("init_empty_desc failed!! err:%d\n",err);
    }
    
    hipStream_t stream1, stream2;
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking); 
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    receive<<<1,RINGS, 0, stream1>>>(rx_desc_base_virt, rdt_reg);
    send<<<1,RINGS, 0, stream2>>>(tx_desc_base_virt, tdt_reg);
    

    printf("Press ENTER key to terminate (Currently not working)\n");
    getchar(); 
    printf("stop\n");

    hipPointerAttribute_t attrs;
    hipPointerGetAttributes(&attrs, d_pointer);
    unpin_mem((uint64_t) attrs.devicePointer);
    hipFree(&d_pointer);
    hipHostUnregister((void*)rdt_reg);
    hipHostUnregister((void*)tdt_reg);
    return 0;
}
